#include "hip/hip_runtime.h"
#pragma warning(disable : 4710)

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef _WIN32
    #define _AMD64_ // architecture
    #define WIN32_LEAN_AND_MEAN
    #define WIN32_EXTRA_MEAN
#endif

#include <errhandlingapi.h>
#include <fileapi.h>
#include <handleapi.h>
#include <sal.h>

// user defined datatypes that conflict with wingdi.h data type are prefixed with a double underscore to avoid conflicts.

static inline char* open(_In_ const wchar_t* const file_name, _Out_ uint64_t* const nread_bytes) {
    *nread_bytes    = 0;
    void *   handle = NULL, *buffer = NULL;
    uint32_t nbytes = 0;

    handle          = CreateFileW(file_name, GENERIC_READ, 0, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_READONLY, NULL);

    if (handle != INVALID_HANDLE_VALUE) {
        LARGE_INTEGER file_size;
        if (!GetFileSizeEx(handle, &file_size)) {
            fwprintf_s(stderr, L"Error %lu in GetFileSizeEx\n", GetLastError());
            return NULL;
        }

        // add an extra megabyte to the buffer, for safety.
        size_t buffsize = file_size.QuadPart + (1024U * 1024);

        // caller is responsible for freeing this buffer.
        buffer          = malloc(buffsize);
        if (buffer) {
            if (ReadFile(handle, buffer, buffsize, (LPDWORD) &nbytes, NULL)) {
                *nread_bytes = nbytes;
                return (char*) buffer;
            } else {
                fwprintf_s(stderr, L"Error %lu in ReadFile\n", GetLastError());
                CloseHandle(handle);
                free(buffer);
                return NULL;
            }
        } else {
            fputws(L"Memory allocation error: malloc returned NULL", stderr);
            CloseHandle(handle);
            return NULL;
        }
    } else {
        fwprintf_s(stderr, L"Error %lu in CreateFileW\n", GetLastError());
        return NULL;
    }
}

__global__ void GetCharCountKernel(_In_ const char* const buffer, _In_ const size_t* length, _Inout_ size_t* const freqs) {
    const char thread_id = threadIdx.x;
    assert(thread_id < 256);
    size_t count = 0;
    for (size_t i = 0; i < *length; ++i) count += (buffer[i] == thread_id);
    freqs[thread_id] = count;
    printf("charcter: %c, count %zd\n", thread_id, count);
    return;
}

int main(void) {
    size_t            frequencies[256] = { 0 };

    size_t            nbytes           = 0;
    const char* const buffer           = open(L"./moby_dick.txt", &nbytes);
    if (!buffer) exit(100);

    char*   device_buffer = NULL;
    size_t* device_freqs  = NULL;
    size_t* device_nbytes = 0;
    hipMalloc(&device_nbytes, sizeof(size_t));
    hipMemcpy(device_buffer, buffer, nbytes, hipMemcpyHostToDevice);
    hipMalloc(&device_freqs, 256 * sizeof(size_t));
    hipMemcpy(device_nbytes, &nbytes, sizeof nbytes, hipMemcpyHostToDevice);

    GetCharCountKernel<<<2, 256>>>(device_buffer, device_nbytes, device_freqs);

    hipDeviceSynchronize();
    hipMemcpy(frequencies, device_freqs, 256 * sizeof(size_t), hipMemcpyDeviceToHost);
    hipFree(device_freqs);
    hipFree(device_buffer);
    hipFree(device_nbytes);
    hipDeviceReset();

    for (size_t i = 0; i < 256; ++i) wprintf_s(L"Frequency of character %c is %10zu.\n", i, frequencies[i]);

    for (size_t i = 0; i < nbytes; ++i) frequencies[buffer[i]]++;
    free((void*) buffer);

    for (size_t i = 0; i < 256; ++i) wprintf_s(L"Frequency of character %C is %10zu.\n", i, frequencies[i]);
    return EXIT_SUCCESS;
}