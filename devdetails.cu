#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#pragma comment(lib, "cuda.lib") // the functions used here come from the CUDA Driver API!!
// linker needs to link the driver API instead of the runtime API!

int main(void) {
    hipInit(NULL);

    int ndevices {};
    hipGetDeviceCount(&ndevices);
    if (ndevices > 1) exit(-1);

    char     device_name[256] {};

    hipDevice_t device;
    hipDeviceGet(&device, 0); // we don't have multiple GPUs here
    hipDeviceGetName(device_name, __crt_countof(device_name), device);

    std::cout << "There are " << ndevices << " NVIDIA CUDA capable GPU's on this machine\n";
    std::cout << "Name of the GPU is " << device_name << std::endl;

    return EXIT_SUCCESS;
}
