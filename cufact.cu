#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

const size_t    SIZE    = 1000;
const size_t    MEMSIZE = SIZE * sizeof(int32_t);

__global__ void Kernel(_In_ const int32_t* const input, _Inout_ long double* const output) {
    const int32_t tindex = threadIdx.x + threadIdx.y + threadIdx.z;
    if (input[tindex] <= 0) {
        output[tindex] = 1;
        return;
    }
    long double result = 1.0L;
    for (int64_t i = 1; i <= input[tindex]; ++i) result *= i;
    output[tindex] = result;
    return;
}

int main(void) {
    srand(time(NULL));

    int32_t* const     randoms     = (int32_t*) malloc(MEMSIZE);
    long double* const results     = (long double*) malloc(SIZE * sizeof(long double));
    int32_t*           dev_rands   = NULL;
    long double*       dev_results = NULL;

    if (!randoms || !results) {
        fputws(L"malloc failed!", stderr);
        goto FREE;
    }

    if (hipSuccess != hipMalloc((void**) &dev_rands, MEMSIZE)) {
        fwprintf_s(stderr, L"hipMalloc failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }
    if (hipSuccess != hipMalloc((void**) &dev_results, SIZE * sizeof(long double))) {
        fwprintf_s(stderr, L"hipMalloc failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }

    for (size_t i = 0; i < SIZE; ++i) randoms[i] = rand() % 20;

    if (hipSuccess != hipMemcpy(dev_rands, randoms, MEMSIZE, hipMemcpyHostToDevice)) {
        fwprintf_s(stderr, L"hipMemcpy failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }

    Kernel<<<1, SIZE>>>(dev_rands, dev_results);
    hipDeviceSynchronize();

    if (hipGetLastError() != hipSuccess) {
        fputws(L"Kernel execution failed!", stderr);
        goto ERROR;
    }

    if (hipSuccess != hipMemcpy(results, dev_results, SIZE * sizeof(long double), hipMemcpyDeviceToHost)) {
        fwprintf_s(stderr, L"hipMemcpy failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }

    for (size_t i = 0; i < SIZE; ++i) wprintf_s(L"Factorial of %4d is %12.5Lf\n", randoms[i], results[i]);
    hipFree(dev_rands);
    hipFree(dev_results);
    free(randoms);
    free(results);

    return EXIT_SUCCESS;

ERROR:
    hipFree(dev_rands);
    hipFree(dev_results);

FREE:
    free(randoms);
    free(results);
    return EXIT_FAILURE;
}