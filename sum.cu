#include "hip/hip_runtime.h"
#include <memory.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 400LLU

__global__ void Kernel(_In_ const uint64_t* const input_0, _In_ const uint64_t* const input_1, _Inout_ uint64_t* const output) {
    const int thread_index = threadIdx.x + threadIdx.y + threadIdx.z;
    output[thread_index]   = input_0[thread_index] + input_1[thread_index];
    return;
}

int main(void) {
    srand(time(NULL));
    const size_t    MEMSIZE = SIZE * sizeof(uint64_t);
    uint64_t *      dev_0 = NULL, *dev_1 = NULL, *dev_results = NULL; // these pointers need to be modifiable, not constant pointers
    // these are meant to be modified my hipMalloc()!!

    uint64_t* const randoms_0 = (uint64_t*) malloc(MEMSIZE);
    uint64_t* const randoms_1 = (uint64_t*) malloc(MEMSIZE);
    uint64_t* const device_   = (uint64_t*) malloc(MEMSIZE);

    if (!randoms_0 || !randoms_1 || !device_) {
        fputws(L"malloc returned NULL!", stderr);
        goto FREE;
    }

    for (size_t i = 0; i < SIZE; ++i) {
        randoms_0[i] = rand();
        randoms_1[i] = rand();
    }

    if (hipSuccess != hipMalloc((void**) &dev_0, MEMSIZE)) {
        fwprintf_s(stderr, L"hipMalloc failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }
    if (hipSuccess != hipMalloc((void**) &dev_1, MEMSIZE)) {
        fwprintf_s(stderr, L"hipMalloc failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }
    if (hipSuccess != hipMalloc((void**) &dev_results, MEMSIZE)) {
        fwprintf_s(stderr, L"hipMalloc failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }

    if (hipSuccess != hipMemcpy(dev_0, randoms_0, MEMSIZE, hipMemcpyHostToDevice)) {
        fwprintf_s(stderr, L"hipMemcpy failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }
    if (hipSuccess != hipMemcpy(dev_1, randoms_1, MEMSIZE, hipMemcpyHostToDevice)) {
        fwprintf_s(stderr, L"hipMemcpy failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }

    Kernel<<<1, SIZE>>>(dev_0, dev_1, dev_results);
    hipDeviceSynchronize();

    if (hipGetLastError() != hipSuccess) {
        fputws(L"Kernel execution failed!", stderr);
        goto ERROR;
    }
    if (hipSuccess != hipMemcpy(device_, dev_results, MEMSIZE, hipMemcpyDeviceToHost)) {
        fwprintf_s(stderr, L"hipMemcpy failed @ __LINE__ %3d\n", __LINE__ - 1);
        goto ERROR;
    }

    for (size_t i = 0; i < SIZE; ++i) wprintf_s(L"Host sum: %8lld, Device sum: %8lld\n", randoms_0[i] + randoms_1[i], device_[i]);
    hipFree(dev_0);
    hipFree(dev_1);
    hipFree(dev_results);
    free(randoms_0);
    free(randoms_1);
    free(device_);

    return EXIT_SUCCESS;

ERROR:
    hipFree(dev_0);
    hipFree(dev_1);
    hipFree(dev_results);

FREE:
    free(randoms_0);
    free(randoms_1);
    free(device_);

    return EXIT_FAILURE;
}