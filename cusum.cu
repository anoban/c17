#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>

// cannot push an MX330 above 2 gigs :(

#define T          1024LLU   // number of threads
#define N          349525LLU // numbers each thread has to sum
#define memsize(x) (N * T * sizeof(x))

__global__ static void kernel(_In_ const double* const nums, _Inout_ double* const results) {
    const unsigned index        = threadIdx.x + threadIdx.y + threadIdx.z;
    const unsigned start_offset = index * N;
    double         res          = 0.00;
    for (size_t i = start_offset; i < start_offset + N; ++i) res += nums[i];
    results[index] = res;
}

__global__ static void reduce(_In_count_(nnums) const double* const nums, _In_ const size_t nnums, _Inout_ double* const result) {
    double res = 0.00;
    for (size_t i = 0; i < nnums; ++i) res += nums[i];
    *result = res;
}

int wmain() {
    srand(time(nullptr));
    double* const numbers = (double*) malloc(memsize(double));
    if (!numbers) { }

    double*     dnumbers    = nullptr;
    double*     dthreadsums = nullptr;
    double*     dgrandsum   = nullptr;
    long double hsum = 0.000L, dsum = 0.000L;
    hipError_t status = hipSuccess;

    // fill the array with real numbers between range 0 and 1
    for (unsigned i = 0; i < N; ++i) hsum += numbers[i] = rand() / (double) RAND_MAX;

    status = hipMalloc(&dnumbers, memsize(double));
    if (status != hipSuccess) { }

    status = hipMemcpy(dnumbers, numbers, memsize(double), hipMemcpyHostToDevice);
    if (status != hipSuccess) { }

    status = hipMalloc(&dthreadsums, T * sizeof(double));
    if (status != hipSuccess) { }

    kernel<<<1, T>>>(dnumbers, dthreadsums);
    status = hipGetLastError();
    if (status != hipSuccess) { }

    status = hipDeviceSynchronize();
    if (status != hipSuccess) { }

    status = hipMalloc(&dgrandsum, sizeof(double));
    if (status != hipSuccess) { }

    reduce<<<1, 1>>>(dthreadsums, T, dgrandsum);
    status = hipGetLastError();
    if (status != hipSuccess) { }

    status = hipMemcpy(&dsum, dgrandsum, sizeof(double), hipMemcpyDeviceToHost);
    if (status != hipSuccess) { }

    hipFree(dnumbers);
    hipFree(dthreadsums);
    hipFree(dgrandsum);

    wprintf_s(L"host :: %Lf, device :: %Lf\n", hsum, dsum); // :)

    return EXIT_SUCCESS;
}
