#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdlib>
#include <execution>
#include <iostream>
#include <numeric>
#include <vector>

#ifdef RAND_MAX
    #undef RAND_MAX
    #define RAND_MAX 100LLU
#endif

__global__ void factkernel(_In_ long double* const d_factorials, _In_ const int32_t* const d_randoms, _In_ const size_t d_size) {
    long double result { 1.000000 };
    const auto  dev_thread = threadIdx.x + threadIdx.y + threadIdx.z;
    for (size_t i = d_randoms[dev_thread]; i > 0; --i) result *= i;
    d_factorials[dev_thread] = result;
    return;
}

constexpr size_t NRANDOMS { 100'000'000 };
constexpr size_t SIZERANDOMS { NRANDOMS * sizeof(int32_t) }; // 400,000,000 bytes (about 38 MiBs)

auto             main(void) -> int {
    srand(time(nullptr));
    std::vector<int32_t> randoms(NRANDOMS);
    std::for_each(randoms.begin(), randoms.end(), [](int32_t& e) {
        e = rand();
        return;
    });

    // store the sum for later comparison
    const auto sum = std::reduce(std::execution::par, randoms.cbegin(), randoms.cend(), 0LLU, std::plus {});

    // device operations
    int32_t*   device_randoms { nullptr };
    hipMalloc(&device_randoms, SIZERANDOMS);
    hipMemcpy(device_randoms, randoms.data(), SIZERANDOMS, hipMemcpyHostToDevice);
    return EXIT_SUCCESS;
}
